#include "hip/hip_runtime.h"
#include "ExtractROIs.cuh"

// CUDA�ںˣ����ڶ��ROI�ü�ͼ�� (2D Kernel)
__global__ void extractRoiKernel(const unsigned char* d_image, unsigned char* d_output, roi_s* d_rois, int roi_count, int width, int height, int channels) {
    // �����̵߳�ȫ������
    int tx = threadIdx.x + blockIdx.x * blockDim.x; // ��������
    int ty = threadIdx.y + blockIdx.y * blockDim.y; // ��������
    // ȷ����������Ч��Χ��
    if (tx < roi_count) {
        // ��ȡ��ǰ ROI
        roi_s roi = d_rois[tx];
        // ����� ROI ��λ�úʹ�С
        int roi_x = roi.x;
        int roi_y = roi.y;
        int roi_width = roi.width;
        int roi_height = roi.height;
        // ���� ROI ������ȡͼ������
        for (int i = 0; i < roi_height; i++) {
            for (int j = 0; j < roi_width; j++) {
                int global_x = roi_x + j;  // ��ǰ���ص�ȫ�� x ����
                int global_y = roi_y + i;  // ��ǰ���ص�ȫ�� y ����
                // ���������Ƿ���ͼ��Χ��
                if (global_x < width && global_y < height) {
                    int image_index = (global_y * width + global_x) * channels;  // ͼ�������
                    int output_index = (tx * roi_width * roi_height + i * roi_width + j) * channels;  // ���������
                    // ��ȡÿ��ͨ��������ֵ
                    for (int c = 0; c < channels; c++) {
                        d_output[output_index + c] = d_image[image_index + c];
                    }
                }
            }
        }
    }
}

ExtractROIs::ExtractROIs(int img_width, int img_height, int img_channel, int roi_num,int roi_width, int roi_height, int device_id)
    : img_width_{ img_width }, 
      img_height_{ img_height },
      img_channel_{ img_channel },
      roi_num_{ roi_num },
      roi_width_{ roi_width },
      roi_height_{ roi_height },
      device_id_{device_id} {
    // ����ͼ�������ڴ�
    hipMalloc(&d_inputImage, img_width * img_height * img_channel * sizeof(unsigned char));
    hipMalloc(&d_outputImage, roi_num * roi_width * roi_height * img_channel * sizeof(unsigned char));  // ÿ�� ROI �����С����
    hipMalloc(&d_rois, roi_num * sizeof(roi_s));
}

ExtractROIs::~ExtractROIs() {
    // �ͷ� GPU �ڴ�
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    hipFree(d_rois);
}

std::vector<roi_s> ExtractROIs::get_split_overlap_rois(int imgWidth, int imgHeight, int startX, int startY, int roiWidth, int roiHeight, int stepX, int stepY) {
    std::vector<roi_s> rois;
    // ���� ROI ������
    for (int y = startY; y + roiHeight <= imgHeight; y += stepY) {
        for (int x = startX; x + roiWidth <= imgWidth; x += stepX) {
            rois.push_back(roi_s{ x,y,roiWidth ,roiHeight });
        }
    }
    return rois;
}

void ExtractROIs::extract_rois_cpu(const unsigned char* buffer, std::vector<roi_s> roi_region, std::vector<cv::Mat>& vec_result_rois_) {
    std::vector<char> vdata(buffer, buffer + img_width_ * img_height_ * img_channel_ * sizeof(unsigned char));
    cv::Mat img = imdecode(cv::Mat(vdata), cv::IMREAD_UNCHANGED);
    vec_result_rois_.clear();
    for (size_t i = 0; i < roi_num_; ++i) {
        vec_result_rois_.push_back(img(cv::Rect(roi_region[i].x, roi_region[i].y, roi_region[i].width, roi_region[i].height)));
    }
}

void ExtractROIs::extract_rois_gpu(const unsigned char* buffer, std::vector<roi_s> roi_region, std::vector<cv::Mat>& vec_result_rois_) {
    vec_result_rois_.clear();
    // ��ͼ�����ݺ� ROI ���ݴ��������䵽 GPU
    hipMemcpy(d_inputImage, buffer, img_width_ * img_height_ * img_channel_ * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_rois, roi_region.data(), roi_region.size() * sizeof(roi_s), hipMemcpyHostToDevice);

    // ���� CUDA �ں˽��вü�
    // ȷ��ÿ���߳̿鴦��һ��ROI
    dim3 blockSize(32, 16);
    dim3 gridSize((roi_region.size() + blockSize.x - 1) / blockSize.x, 1);
    // ���� CUDA �˺���
    extractRoiKernel <<<gridSize, blockSize>>> (d_inputImage, d_outputImage, d_rois, roi_region.size(), img_width_, img_height_, img_channel_);
    // �ȴ� CUDA �˺���ִ�����
    hipDeviceSynchronize();

    // ��� CUDA �ں�ִ���Ƿ�ɹ�
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        //std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        //return -1;
    }

    // �� GPU �����ü������ CPU
    std::vector<uchar> outputImage(roi_region.size() * roi_width_ * roi_height_);       // �������ߴ��� 100x100
    hipMemcpy(outputImage.data(), d_outputImage, roi_region.size() * roi_width_ * roi_height_ * img_channel_ * sizeof(unsigned char), hipMemcpyDeviceToHost);

    for (int i = 0; i < roi_region.size(); ++i) {
        if(img_channel_ ==1)
            vec_result_rois_.push_back(cv::Mat(roi_region[i].height, roi_region[i].width, CV_8UC1, outputImage.data() + i * roi_region[i].width * roi_region[i].height));
        else if (img_channel_ == 3) {
            vec_result_rois_.push_back(cv::Mat(roi_region[i].height, roi_region[i].width, CV_8UC3, outputImage.data() + i * roi_region[i].width * roi_region[i].height));
        }
    }
}


/*
void ExtractROIs::output_result() {

}
*/

